#include "hip/hip_runtime.h"
#include <iostream>

using namespace std;

#define Threads 3
#define Blocks  4
#define N Threads*Blocks

__global__  // GPU function
void add(int *a, int *b, int n)
{
    // Get ID of thread being executed
	int tid = threadIdx.x + blockIdx.x * blockDim.x;
	// if the thread id is less than the number of loops required
	if (tid < n)
	    // Add them together
		b[tid] += a[tid];
	// Notice there is no return statement
}

int main(void)
{
    // Calculate memory size
	int memSize = N*sizeof(int);

    // Initialize host (CPU) memory
	int *h_a, *h_b;
	h_a = (int*)malloc(memSize);
	h_b = (int*)malloc(memSize);

    // Initialize device (GPU) memory
	int *d_a, *d_b;
	hipMalloc((void**)&d_a, memSize);
	hipMalloc((void**)&d_b, memSize);

    // Add some values to host arrays a and b to sum.
	for (int i = 0; i < N; i++) {
		h_a[i] = i;
		h_b[i] = i*i;
	}

    // Send host (CPU) memory to device (GPU)
	hipMemcpy(d_a, h_a, memSize, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, memSize, hipMemcpyHostToDevice);

    // Run function add() on device (GPU)
	add<<<Blocks, Threads>>>(d_a, d_b, N);

    // Make sure all threads on GPU finish
	hipDeviceSynchronize();

    // Send device (GPU) memory back to host(CPU)
	hipMemcpy(h_b, d_b, memSize, hipMemcpyDeviceToHost);

    // Print output from device (GPU)
	for (int i = 0; i < N; i++)
		cout << h_b[i] << "\n";

	// Free host (CPU) memory
	free(h_a);
	free(h_b);
	
	// Free device (GPU) memory
	hipFree(d_a);
	hipFree(d_b);

    // Exit with success!
	return 1;
}

